
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

__global__
void vecAddKernel(float* d_A, float* d_B, float* d_C, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        d_C[i] = d_A[i] + d_B[i];
    }
}

void vecAdd(float* h_A, float* h_B, float* h_C, int n)
{
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_C, size);

    vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_B);
}

int main(void)
{
    int n = 100;
    float* h_A = new float[n];
    float* h_B = new float[n];
    float* h_C = new float[n];

    for (int i = 0; i < n; ++i)
    {
        h_A[i] = i;
        h_B[i] = i;
    }

    vecAdd(h_A, h_B, h_C, n);

    for (int i = 0; i < n; ++i)
    {
        std::cout << h_C[i] << " ";
    }
    std::cout << "\n";

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
	
	return 0;
}